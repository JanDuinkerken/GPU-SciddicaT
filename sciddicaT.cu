#include "hip/hip_runtime.h"
#include "util.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5
// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

// ----------------------------------------------------------------------------
// Tiled Halo Cell parameters
// ----------------------------------------------------------------------------
#define MAX_MASK_WIDTH 3
#define T_WIDTH 30
#define T_BLOCK_WIDTH (T_WIDTH + MAX_MASK_WIDTH - 1)
#define T_BUFF_SIZE (T_BLOCK_WIDTH * T_BLOCK_WIDTH)

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rows, columns, n, i, j, value) \
  ((M)[(((n) * (rows) * (columns)) + ((i) * (columns)) + (j))] = (value))
#define BUF_GET(M, rows, columns, n, i, j) \
  (M[(((n) * (rows) * (columns)) + ((i) * (columns)) + (j))])

// ----------------------------------------------------------------------------
// Inline error checking
// ----------------------------------------------------------------------------
#define gpuErrchk(ans)                    \
  {                                       \
    gpuAssert((ans), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char *path, int &nrows, int &ncols,
                    /*double &xllcorner, double &yllcorner, double &cellsize,*/
                    double &nodata)
{
  FILE *f;

  if ((f = fopen(path, "r")) == 0)
  {
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  // Reading the header
  char str[STRLEN];
  fscanf(f, "%s", &str);
  fscanf(f, "%s", &str);
  ncols = atoi(str); // ncols
  fscanf(f, "%s", &str);
  fscanf(f, "%s", &str);
  nrows = atoi(str); // nrows
  fscanf(f, "%s", &str);
  fscanf(f, "%s", &str); // xllcorner = atof(str);  //xllcorner
  fscanf(f, "%s", &str);
  fscanf(f, "%s", &str); // yllcorner = atof(str);  //yllcorner
  fscanf(f, "%s", &str);
  fscanf(f, "%s", &str); // cellsize = atof(str);   //cellsize
  fscanf(f, "%s", &str);
  fscanf(f, "%s", &str);
  nodata = atof(str); // NODATA_value
}

bool loadGrid2D(double *M, int rows, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f)
  {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rows; i++)
    for (int j = 0; j < columns; j++)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rows, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rows; i++)
  {
    for (int j = 0; j < columns; j++)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

double *addLayer2D(int rows, int columns)
{
  double *tmp;
  gpuErrchk(hipMallocManaged(&tmp, sizeof(double) * rows * columns));

  if (!tmp)
    return NULL;
  return tmp;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop (Does not benefit of a tiled implementation)
// ----------------------------------------------------------------------------
__global__ void sciddicaTSimulationInitKernel(int r, int c, double *Sz,
                                              double *Sh)
{
  int row_index = threadIdx.y + blockDim.y * blockIdx.y;
  int col_index = threadIdx.x + blockDim.x * blockIdx.x;
  int row_stride = blockDim.y * gridDim.y;
  int col_stride = blockDim.x * gridDim.x;

  for (int row = row_index + 1; row < r - 1; row += row_stride)
  {
    for (int col = col_index + 1; col < c - 1; col += col_stride)
    {
      double z, h;
      h = GET(Sh, c, row, col);

      if (h > 0.0)
      {
        z = GET(Sz, c, row, col);
        SET(Sz, c, row, col, z - h);
      }
    }
  }
}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------

// This kernel does not benefit from a tiled implementation
__global__ void sciddicaTResetFlowsKernel(int r, int c, double nodata, double *Sf)
{
  int row_index = blockDim.y * blockIdx.y + threadIdx.y;
  int row_stride = blockDim.y * gridDim.y;
  int col_index = blockDim.x * blockIdx.x + threadIdx.x;
  int col_stride = blockDim.x * gridDim.x;

  for (int row = row_index + 1; row < r - 1; row += row_stride)
  {
    for (int col = col_index + 1; col < c - 1; col += col_stride)
    {
      BUF_SET(Sf, r, c, 0, row, col, 0.0);
      BUF_SET(Sf, r, c, 1, row, col, 0.0);
      BUF_SET(Sf, r, c, 2, row, col, 0.0);
      BUF_SET(Sf, r, c, 3, row, col, 0.0);
    }
  }
}

// This kernel benefits from a tiled implementation
__global__ void sciddicaTFlowsComputationSimpleKernel(int r, int c, double nodata, int *Xi, int *Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon)
{
  int col_index = threadIdx.x + blockDim.x * blockIdx.x;
  int row_index = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;

  for (int row = row_index + 1; row < r - 1; row += row_stride)
  {
    for (int col = col_index + 1; col < c - 1; col += col_stride)
    {
      m = GET(Sh, c, row, col) - p_epsilon;
      u[0] = GET(Sz, c, row, col) + p_epsilon;
      z = GET(Sz, c, row + Xi[1], col + Xj[1]);
      h = GET(Sh, c, row + Xi[1], col + Xj[1]);
      u[1] = z + h;
      z = GET(Sz, c, row + Xi[2], col + Xj[2]);
      h = GET(Sh, c, row + Xi[2], col + Xj[2]);
      u[2] = z + h;
      z = GET(Sz, c, row + Xi[3], col + Xj[3]);
      h = GET(Sh, c, row + Xi[3], col + Xj[3]);
      u[3] = z + h;
      z = GET(Sz, c, row + Xi[4], col + Xj[4]);
      h = GET(Sh, c, row + Xi[4], col + Xj[4]);
      u[4] = z + h;

      do
      {
        again = false;
        average = m;
        cells_count = 0;

        for (n = 0; n < 5; ++n)
          if (!eliminated_cells[n])
          {
            average += u[n];
            ++cells_count;
          }

        if (cells_count != 0)
          average /= cells_count;

        for (n = 0; n < 5; ++n)
          if ((average <= u[n]) && (!eliminated_cells[n]))
          {
            eliminated_cells[n] = true;
            again = true;
          }
      } while (again);

      if (!eliminated_cells[1])
        BUF_SET(Sf, r, c, 0, row, col, (average - u[1]) * p_r);
      if (!eliminated_cells[2])
        BUF_SET(Sf, r, c, 1, row, col, (average - u[2]) * p_r);
      if (!eliminated_cells[3])
        BUF_SET(Sf, r, c, 2, row, col, (average - u[3]) * p_r);
      if (!eliminated_cells[4])
        BUF_SET(Sf, r, c, 3, row, col, (average - u[4]) * p_r);
    }
  }
}

__global__ void sciddicaTFlowsComputationKernel(int r, int c, double nodata, int *Xi, int *Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon)
{
  int col_index = threadIdx.x + blockDim.x * blockIdx.x;
  int row_index = threadIdx.y + blockDim.y * blockIdx.y;

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z = 0, h = 0;

  __shared__ double Sz_ds[T_WIDTH][T_WIDTH];
  __shared__ double Sh_ds[T_WIDTH][T_WIDTH];

  Sz_ds[threadIdx.y][threadIdx.x] = GET(Sz, c, row_index, col_index);
  Sh_ds[threadIdx.y][threadIdx.x] = GET(Sh, c, row_index, col_index);
  __syncthreads();

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  m = Sh_ds[threadIdx.y][threadIdx.x] - p_epsilon;
  u[0] = Sz_ds[threadIdx.y][threadIdx.x] + p_epsilon;

  int index_x;
  int index_y;

  for (int tmp = 0; tmp < MAX_MASK_WIDTH; tmp++)
  {
    index_y = row_index - (MAX_MASK_WIDTH / 2) + Xi[tmp + 1];
    index_x = col_index - (MAX_MASK_WIDTH / 2) + Xj[tmp + 1];

    if ((index_x >= 0) && (index_x < c) && (index_y >= 0) && (index_y < r))
    {
      if ((index_x >= tile_start_x) && (index_x < next_tile_start_x) && (index_y >= tile_start_y) && (index_y < next_tile_start_y))
      {
        z = Sz_ds[threadIdx.y + Xi[tmp + 1]][threadIdx.x + MAX_MASK_WIDTH / 2 + Xj[tmp + 1]];
        h = Sh_ds[threadIdx.y + Xi[tmp + 1]][threadIdx.x + MAX_MASK_WIDTH / 2 + Xj[tmp + 1]];
      }
      else
      {
        z = GET(Sz, c, index_y, index_x);
        h = GET(Sh, c, index_y, index_x);
      }
    }
    u[tmp + 1] = z + h;
  }

  do
  {
    again = false;
    average = m;
    cells_count = 0;

    for (n = 0; n < 5; n++)
      if (!eliminated_cells[n])
      {
        average += u[n];
        cells_count++;
      }

    if (cells_count != 0)
      average /= cells_count;

    for (n = 0; n < 5; n++)
      if ((average <= u[n]) && (!eliminated_cells[n]))
      {
        eliminated_cells[n] = true;
        again = true;
      }
  } while (again);

  if (!eliminated_cells[0])
    BUF_SET(Sf, r, c, 0, row_index, col_index, (average - u[0]) * p_r);

  if (!eliminated_cells[1])
    BUF_SET(Sf, r, c, 1, row_index, col_index, (average - u[1]) * p_r);

  if (!eliminated_cells[2])
    BUF_SET(Sf, r, c, 2, row_index, col_index, (average - u[2]) * p_r);

  if (!eliminated_cells[3])
    BUF_SET(Sf, r, c, 3, row_index, col_index, (average - u[3]) * p_r);
}

// This kernel benefits from a tiled implementation
__global__ void sciddicaTWidthUpdateSimpleKernel(int r, int c, double nodata, int *Xi,
                                           int *Xj, double *Sz, double *Sh, double *Sf)
{
  int row_index = threadIdx.y + blockDim.y * blockIdx.y;
  int col_index = threadIdx.x + blockDim.x * blockIdx.x;
  int row_stride = blockDim.y * gridDim.y;
  int col_stride = blockDim.x * gridDim.x;

  for (int row = row_index + 1; row < r - 1; row += row_stride)
  {
    for (int col = col_index + 1; col < c - 1; col += col_stride)
    {
      double h_next;
      h_next = GET(Sh, c, row, col);
      h_next +=
          BUF_GET(Sf, r, c, 3, row + Xi[1], col + Xj[1]) - BUF_GET(Sf, r, c, 0, row, col);
      h_next +=
          BUF_GET(Sf, r, c, 2, row + Xi[2], col + Xj[2]) - BUF_GET(Sf, r, c, 1, row, col);
      h_next +=
          BUF_GET(Sf, r, c, 1, row + Xi[3], col + Xj[3]) - BUF_GET(Sf, r, c, 2, row, col);
      h_next +=
          BUF_GET(Sf, r, c, 0, row + Xi[4], col + Xj[4]) - BUF_GET(Sf, r, c, 3, row, col);

      SET(Sh, c, row, col, h_next);
    }
  }
}

__global__ void sciddicaTWidthUpdateKernel(int r, int c, double nodata, int *Xi, int *Xj, double *Sz, double *Sh, double *Sf)
{
  int col_index = threadIdx.x + blockDim.x * blockIdx.x;
  int row_index = threadIdx.y + blockDim.y * blockIdx.y;

  double h_next;

  __shared__ double Sf_ds[T_WIDTH * ADJACENT_CELLS][T_WIDTH];

  Sf_ds[threadIdx.y][threadIdx.x] = BUF_GET(Sf, r, c, 0, row_index, col_index);
  Sf_ds[threadIdx.y + T_WIDTH][threadIdx.x] = BUF_GET(Sf, r, c, 1, row_index, col_index);
  Sf_ds[threadIdx.y + T_WIDTH * 2][threadIdx.x] = BUF_GET(Sf, r, c, 2, row_index, col_index);
  Sf_ds[threadIdx.y + T_WIDTH * 3][threadIdx.x] = BUF_GET(Sf, r, c, 3, row_index, col_index);
  __syncthreads();

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  h_next = GET(Sh, c, row_index, col_index);

  for (int tmp = 0; tmp <= MAX_MASK_WIDTH; ++tmp)
  {
    int n_index_x = col_index - (MAX_MASK_WIDTH / 2) + Xj[tmp + 1];
    int n_index_y = row_index - (MAX_MASK_WIDTH / 2) + Xi[tmp + 1];
    if ((n_index_x >= 0) && (n_index_x < c) && (n_index_y >= 0) && (n_index_y < r))
    {
      if ((n_index_x >= tile_start_x) && (n_index_x < next_tile_start_x) && (n_index_y >= tile_start_y) && (n_index_y < next_tile_start_y))
      {
        h_next += Sf_ds[threadIdx.y + T_WIDTH * (MAX_MASK_WIDTH - tmp) + Xi[tmp + 1]][threadIdx.x + Xj[tmp + 1]] - Sf_ds[threadIdx.y + T_WIDTH * tmp][threadIdx.x];
      }
      else
      { // try to get a L2 cache hit (best case, otherwise global memory in DRAM has to be accessed)
        h_next += BUF_GET(Sf, r, c, (MAX_MASK_WIDTH - tmp), n_index_y, n_index_x) - BUF_GET(Sf, r, c, tmp, n_index_y, n_index_x);
      }
    }
  }

  SET(Sh, c, row_index, col_index, h_next); // TODO check calculation results
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rows, cols;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rows, cols, nodata);

  int r = rows; // r: grid rows
  int c = cols; // c: grid columns
  double *Sz;   // Sz: substate (grid) containing the cells' altitude a.s.l.
  double *Sh;   // Sh: substate (grid) containing the cells' flow thickness
  double *Sf;   // Sf: 4 substates containing the flows towards the 4 neighs

  int *Xi;
  int *Xj;

  gpuErrchk(hipMallocManaged(&Xi, sizeof(int) * 5));
  gpuErrchk(hipMallocManaged(&Xj, sizeof(int) * 5));

  // Xj: von Neuman neighborhood row coordinates (see below)
  Xi[0] = 0;
  Xi[1] = -1;
  Xi[2] = 0;
  Xi[3] = 0;
  Xi[4] = 1;
  // Xj: von Neuman neighborhood col coordinates (see below)
  Xj[0] = 0;
  Xj[1] = 0;
  Xj[2] = -1;
  Xj[3] = 1;
  Xj[4] = 0;

  double p_r = P_R;                 // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;     // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); // steps: simulation steps

  dim3 tiled_block_size(T_WIDTH, T_WIDTH, 1); // == T_BUFF_SIZE
  dim3 tiled_grid_size(ceil(rows / T_WIDTH), ceil(cols / T_WIDTH), 1);

  // Not all kernels are going to use a tiled implementation so we keep the normal grid and block size variables
  int n = rows * cols;
  int dim_x = 32;
  int dim_y = 32;
  dim3 block_size(dim_x, dim_y, 1);
  dim3 grid_size(ceil(sqrt(n / (dim_x * dim_y))), ceil(sqrt(n / (dim_x * dim_y))), 1);

  // The adopted von Neuman neighborhood
  // Format: flow_index:cell_label:(row_index,col_index)
  //
  //   cell_label in [0,1,2,3,4]: label assigned to each cell in the
  //   neighborhood flow_index in   [0,1,2,3]: outgoing flow indices in Sf from
  //   cell 0 to the others
  //       (row_index,col_index): 2D relative indices of the cells
  //
  //               |0:1:(-1, 0)|
  //   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
  //               |3:4:( 1, 0)|
  //
  //

  Sz = addLayer2D(r, c); // Allocates the Sz substate grid
  Sh = addLayer2D(r, c); // Allocates the Sh substate grid
  Sf = addLayer2D(ADJACENT_CELLS * r,
                  c); // Allocates the Sf substates grid,
                      //   having one layer for each adjacent cell

  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);    // Load Sz from file
  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]); // Load Sh from file

  // Apply the init kernel (elementary process) to the whole domain grid
  // (cellular space)
  sciddicaTSimulationInitKernel<<<grid_size, block_size>>>(r, c, Sz, Sh);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  util::Timer cl_timer;
  // simulation loop
  for (int s = 0; s < steps; ++s)
  {
    // Apply the resetFlow kernel to the whole domain
    sciddicaTResetFlowsKernel<<<grid_size, block_size>>>(r, c, nodata, Sf);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    // Apply the FlowComputation kernel to the whole domain
    sciddicaTFlowsComputationKernel<<<tiled_grid_size, tiled_block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    // Apply the WidthUpdate mass balance kernel to the whole domain
    sciddicaTWidthUpdateKernel<<<tiled_grid_size, tiled_block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
  }
  double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
  printf("Elapsed time: %lf [s]\n", cl_time);

  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]); // Save Sh to file

  printf("Releasing memory...\n");
  gpuErrchk(hipFree(Sz));
  gpuErrchk(hipFree(Sh));
  gpuErrchk(hipFree(Sf));

  return 0;
}
